#include"cu_data.h"
#include"data.h"
#include"csolver_gpu.h"
#include<stdio.h>
#include<unistd.h>

//1D const 
float *d_zz;//added by momentum3d
float *d_dz;//added by mode_interaction 
float *d_uabe, *d_uabw, //added by mode_external
 	  *d_ele, *d_elw;
float *d_vabs, *d_vabn, 
	  *d_els, *d_eln;
float *d_dzz, *d_z;//added by mode_internal-->profq

//2D const
float *d_dx, *d_dy;//added by momentum3d 
float *d_aru, *d_arv;//added by momentum3d-->advct
float *d_aamfrz;//added by momentum3d 
float *d_art, *d_cor;//added by mode_external 
float *d_h, *d_fsm;//added by mode_external 
float *d_frz;//added by mode_internal-->bcond(4)


//3D const
float *d_tclim, *d_sclim;//added by mode_internal-->advt1


float *d_relax_aid;//added by mode_internal-->advt1

//2D
float *d_vfluxf, *d_e_atmos;//added by surface_forcing
float *d_swrad;//added by surface_forcing
float *d_dum, *d_dvm;//added by momentum3d 
float *d_dt, *d_d;//added by momentum3d 
float *d_adx2d, *d_ady2d;//added by mode_interaction 
float *d_drx2d, *d_dry2d;//added by mode_interaction 
float *d_aam2d;//added by mode_interaction 
float *d_ua, *d_va;//added by mode_interaction 
float *d_uab, *d_vab;//added by mode_interaction 
float *d_wubot, *d_wvbot;//added by mode_interaction 
float *d_advua, *d_advva;//added by mode_interaction 
float *d_el, *d_egf;//added by mode_interaction 
float *d_utf, *d_vtf;//added by mode_interaction 
float *d_cbc;//added by mode_interaction 

float *d_elb, *d_elf;//added by mode_external
float *d_wusurf, *d_wvsurf;//added by mode_external
float *d_etf;//added by mode_external
float *d_uaf, *d_vaf;//added by mode_external

float *d_utb, *d_vtb;//added by mode_internal
float *d_etb, *d_vfluxb;//added by mode_internal
float *d_tsurf;//added by mode_internal-->advt1
float *d_wtsurf, *d_wssurf;//added by mode_internal-->proft 
float *d_ssurf;//added by mode_internal-->proft

float *d_tbe, *d_sbe, *d_tbw, *d_sbw;//added by mode_internal-->bcond(4)
float *d_tbs, *d_sbs, *d_tbn, *d_sbn;//added by mode_internal-->bcond(4)

float *d_egb;//added by mode_internal-->advu
float *d_et;//added by mode_internal-->kernel_4

float *d_uab_mean, *d_vab_mean;//added by store_mean
float *d_elb_mean;//added by store_mean
float *d_wusurf_mean, *d_wvsurf_mean;//added by store_mean
float *d_wtsurf_mean, *d_wssurf_mean;//added by store_mean

float *d_usrf_mean, *d_vsrf_mean;//add by store_surf_mean
float *d_elsrf_mean;//add by store_surf_mean 
float *d_uwsrf_mean, *d_vwsrf_mean;//add by store_surf_mean
float *d_uwsrf, *d_vwsrf;//add by store_surf_mean
float *d_utf_mean, *d_vtf_mean;//add by store_surf_mean
float *d_celg_mean, *d_ctsurf_mean;//add by store_surf_mean
float *d_celg, *d_ctsurf;//add by store_surf_mean
float *d_cpvf_mean, *d_cjbar_mean;//add by store_surf_mean
float *d_cpvf, *d_cjbar;//add by store_surf_mean
float *d_cadv_mean, *d_cten_mean;//add by store_surf_mean
float *d_cadv, *d_cten;//add by store_surf_mean
float *d_ctbot_mean;//add by store_surf_mean
float *d_ctbot;//add by store_surf_mean

float *d_ctot, *d_totx, *d_toty;//add by mode_external-->vort

//3D
float *d_w;//added by surface_forcing
float *d_u, *d_v;//added by momentum3d 
float *d_ub, *d_vb;//added by momentum3d 
float *d_aam, *d_rho, *d_rmean;//added by momentum3d 
float *d_advx, *d_advy;//added by momentum3d 
float *d_drhox, *d_drhoy;//added by momentum3d 

float *d_q2b, *d_q2;//added by mode_internal 
float *d_q2lb, *d_q2l;//added by mode_internal 
float *d_uf, *d_vf;//added by mode_internal 
float *d_kq, *d_l;//added by mode_internal-->profq 
float *d_t, *d_s;//added by mode_internal-->profq 
float *d_km, *d_kh;//added by mode_internal-->profq 
float *d_tb, *d_sb; //added by mode_internal-->advt1

float *d_tobw, *d_sobw, //added by mode_internal-->bcond(4)
	  *d_tobe, *d_sobe,
	  *d_tobs, *d_sobs,
	  *d_tobn, *d_sobn;

float *d_u_mean, *d_v_mean, *d_w_mean;//added by store_mean
float *d_t_mean, *d_s_mean;//added by store_mean
float *d_rho_mean, *d_kh_mean, *d_km_mean;//added by store_mean
float *d_ustks, *d_vstks;//added by store_mean

float *d_xstks_mean, *d_ystks_mean;//add by store_surf_mean
float *d_xstks, *d_ystks;//add by store_surf_mean

float *d_3d_tmp0, *d_3d_tmp1, 
	  *d_3d_tmp2, *d_3d_tmp3;

float *d_3d_tmp4, *d_3d_tmp5, 
	  *d_3d_tmp6, *d_3d_tmp7,
	  *d_3d_tmp8, *d_3d_tmp9,
	  *d_3d_tmp10, *d_3d_tmp11;

float *d_2d_tmp0, *d_2d_tmp1;

float *d_2d_tmp2, *d_2d_tmp3,
	  *d_2d_tmp4, *d_2d_tmp5,
	  *d_2d_tmp6, *d_2d_tmp7,
	  *d_2d_tmp8, *d_2d_tmp9,
	  *d_2d_tmp10, *d_2d_tmp11,
	  *d_2d_tmp12, *d_2d_tmp13;

float *d_1d_ny_tmp0, *d_1d_ny_tmp1,
	  *d_1d_ny_tmp2, *d_1d_ny_tmp3;

//////////////////////////////////////////////
//data copy back to host and then mpi
float *h_1d_nx_tmp0, *h_1d_nx_tmp1,
	  *h_1d_nx_tmp2, *h_1d_nx_tmp3;
float *h_1d_ny_tmp0, *h_1d_ny_tmp1,
	  *h_1d_ny_tmp2, *h_1d_ny_tmp3;
//////////////////////////////////////////////

float *d_2d_ny_nz_tmp0, *d_2d_ny_nz_tmp1,
 	  *d_2d_ny_nz_tmp2, *d_2d_ny_nz_tmp3;

float *d_2d_nx_nz_tmp0, *d_2d_nx_nz_tmp1,
	  *d_2d_nx_nz_tmp2, *d_2d_nx_nz_tmp3;


//////////////////////////////////////////////
//data copy back to host and then mpi
float *h_2d_ny_nz_tmp0, *h_2d_ny_nz_tmp1,
 	  *h_2d_ny_nz_tmp2, *h_2d_ny_nz_tmp3;

float *h_2d_nx_nz_tmp0, *h_2d_nx_nz_tmp1,
 	  *h_2d_nx_nz_tmp2, *h_2d_nx_nz_tmp3;
//////////////////////////////////////////////


///////////////////////////////////////
//for the use of cudaIpc
float *d_ctsurf_east, *d_ctsurf_west,
 	  *d_ctbot_east, *d_ctbot_west,
 	  *d_celg_east, *d_celg_west,
 	  *d_cjbar_east, *d_cjbar_west,
 	  *d_cadv_east, *d_cadv_west,
 	  *d_cpvf_east, *d_cpvf_west,
 	  *d_cten_east, *d_cten_west;

/////////////////////////////////////////////
float *d_2d_tmp0_east, *d_2d_tmp0_west,
	  *d_2d_tmp1_east, *d_2d_tmp1_west,
	  *d_2d_tmp2_east, *d_2d_tmp2_west,
	  *d_2d_tmp3_east, *d_2d_tmp3_west,
	  *d_2d_tmp4_east, *d_2d_tmp4_west,
	  *d_2d_tmp5_east, *d_2d_tmp5_west,
	  *d_2d_tmp6_east, *d_2d_tmp6_west,
	  *d_2d_tmp7_east, *d_2d_tmp7_west,
	  *d_2d_tmp8_east, *d_2d_tmp8_west,
	  *d_2d_tmp9_east, *d_2d_tmp9_west,
	  *d_2d_tmp10_east, *d_2d_tmp10_west,
	  *d_2d_tmp11_east, *d_2d_tmp11_west,
	  *d_2d_tmp12_east, *d_2d_tmp12_west,
	  *d_2d_tmp13_east, *d_2d_tmp13_west;

float *d_2d_tmp0_south, *d_2d_tmp0_north,
	  *d_2d_tmp1_south, *d_2d_tmp1_north;
/////////////////////////////////////////////

float *d_totx_east, *d_totx_west,
	  *d_toty_east, *d_toty_west;

/////////////////////////////////////////////
float *d_3d_tmp0_east, *d_3d_tmp0_west,
	  *d_3d_tmp1_east, *d_3d_tmp1_west,
	  *d_3d_tmp2_east, *d_3d_tmp2_west;

float *d_3d_tmp0_south, *d_3d_tmp0_north,
	  *d_3d_tmp1_south, *d_3d_tmp1_north,
	  *d_3d_tmp2_south, *d_3d_tmp2_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_wubot_east, *d_wubot_west,
	  *d_wubot_east_most, *d_wubot_west_most,
	  *d_wvbot_east, *d_wvbot_west,
	  *d_wvbot_east_most, *d_wvbot_west_most;

float *d_wubot_south, *d_wubot_north,
	  *d_wvbot_south, *d_wvbot_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_aam_east, *d_aam_west;
float *d_aam_south, *d_aam_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_elf_east, *d_elf_west;
float *d_elf_east_most, *d_elf_west_most;
float *d_elf_south, *d_elf_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_uaf_east, *d_uaf_west;
float *d_uaf_east_most, *d_uaf_west_most;
float *d_uaf_south, *d_uaf_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_vaf_east, *d_vaf_west;
float *d_vaf_east_most, *d_vaf_west_most;
float *d_vaf_south, *d_vaf_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_w_east, *d_w_west;
float *d_w_east_most, *d_w_west_most;
float *d_w_south, *d_w_north;
/////////////////////////////////////////////


/////////////////////////////////////////////
float *d_uf_east, *d_uf_west,
	  *d_uf_south, *d_uf_north,
	  *d_vf_east, *d_vf_west,
	  *d_vf_south, *d_vf_north,
	  *d_uf_east_most, *d_uf_west_most,
	  *d_vf_east_most, *d_vf_west_most,
	  *d_kh_east_most, *d_kh_west_most,
	  *d_km_east_most, *d_km_west_most,
	  *d_kq_east_most, *d_kq_west_most;
/////////////////////////////////////////////

hipIpcMemHandle_t handle_ctsurf,
				   handle_ctbot,
				   handle_celg,
				   handle_cjbar,
				   handle_cadv,
				   handle_cpvf,
				   handle_cten,
				   handle_ctsurf_east, handle_ctsurf_west,
			       handle_ctbot_east, handle_ctbot_west,
			       handle_celg_east, handle_celg_west,
			       handle_cjbar_east, handle_cjbar_west,
			       handle_cadv_east, handle_cadv_west,
			       handle_cpvf_east, handle_cpvf_west,
			       handle_cten_east, handle_cten_west;

/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_2d_tmp0, handle_2d_tmp1,
				   handle_2d_tmp2, handle_2d_tmp3,
				   handle_2d_tmp4, handle_2d_tmp5,
				   handle_2d_tmp6, handle_2d_tmp7,
				   handle_2d_tmp8, handle_2d_tmp9,
				   handle_2d_tmp10, handle_2d_tmp11,
				   handle_2d_tmp12, handle_2d_tmp13,
				   handle_2d_tmp0_east, handle_2d_tmp0_west,
				   handle_2d_tmp0_south, handle_2d_tmp0_north,
				   handle_2d_tmp1_east, handle_2d_tmp1_west,
				   handle_2d_tmp1_south, handle_2d_tmp1_north,
				   handle_2d_tmp2_east, handle_2d_tmp2_west,
				   handle_2d_tmp3_east, handle_2d_tmp3_west,
				   handle_2d_tmp4_east, handle_2d_tmp4_west,
				   handle_2d_tmp5_east, handle_2d_tmp5_west,
				   handle_2d_tmp6_east, handle_2d_tmp6_west,
				   handle_2d_tmp7_east, handle_2d_tmp7_west,
				   handle_2d_tmp8_east, handle_2d_tmp8_west,
				   handle_2d_tmp9_east, handle_2d_tmp9_west,
				   handle_2d_tmp10_east, handle_2d_tmp10_west,
				   handle_2d_tmp11_east, handle_2d_tmp11_west,
				   handle_2d_tmp12_east, handle_2d_tmp12_west,
				   handle_2d_tmp13_east, handle_2d_tmp13_west;
/////////////////////////////////////////////////////////////////

hipIpcMemHandle_t handle_totx, handle_toty,
				   handle_totx_east, handle_totx_west,
				   handle_toty_east, handle_toty_west;

/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_3d_tmp0, handle_3d_tmp1, handle_3d_tmp2,
				   handle_3d_tmp0_east, handle_3d_tmp0_west,
				   handle_3d_tmp0_south, handle_3d_tmp0_north,
				   handle_3d_tmp1_east, handle_3d_tmp1_west,
				   handle_3d_tmp1_south, handle_3d_tmp1_north,
				   handle_3d_tmp2_east, handle_3d_tmp2_west,
				   handle_3d_tmp2_south, handle_3d_tmp2_north;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_wubot, handle_wvbot,
				   handle_wubot_east, handle_wubot_west,
				   handle_wubot_south, handle_wubot_north,
				   handle_wubot_east_most, handle_wubot_west_most,
				   handle_wvbot_east, handle_wvbot_west,
				   handle_wvbot_south, handle_wvbot_north,
				   handle_wvbot_east_most, handle_wvbot_west_most;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_aam,
				   handle_aam_east, handle_aam_west,
				   handle_aam_south, handle_aam_north;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_elf,
				   handle_elf_east, handle_elf_west,
				   handle_elf_south, handle_elf_north,
				   handle_elf_east_most, handle_elf_west_most;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_uaf, handle_vaf,
				   handle_uaf_east, handle_uaf_west,
				   handle_uaf_south, handle_uaf_north,
				   handle_vaf_east, handle_vaf_west,
				   handle_vaf_south, handle_vaf_north,
				   handle_uaf_east_most, handle_uaf_west_most,
				   handle_vaf_east_most, handle_vaf_west_most;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_w,
				   handle_w_east, handle_w_west,
				   handle_w_south, handle_w_north,
				   handle_w_east_most, handle_w_west_most;
/////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////
hipIpcMemHandle_t handle_uf, handle_vf,
				   handle_kh, handle_km, handle_kq,
				   handle_uf_east, handle_uf_west,
				   handle_uf_south, handle_uf_north,
				   handle_vf_east, handle_vf_west,
				   handle_vf_south, handle_vf_north,
				   handle_uf_east_most, handle_uf_west_most,
				   handle_vf_east_most, handle_vf_west_most,
				   handle_kh_east_most, handle_kh_west_most,
				   handle_km_east_most, handle_km_west_most,
				   handle_kq_east_most, handle_kq_west_most;
/////////////////////////////////////////////////////////////////



hipStream_t stream[5];

dim3 threadPerBlock(block_i_2D, block_j_2D);
dim3 blockPerGrid((i_size+block_i_2D-1)/block_i_2D,
				  (j_size+block_j_2D-1)/block_j_2D);

dim3 threadPerBlock_inner(block_i_2D, block_j_2D);
dim3 blockPerGrid_inner((i_size-2-64+block_i_2D-1)/block_i_2D,
					    (j_size-2-64+block_j_2D-1)/block_j_2D);

dim3 threadPerBlock_ew_32(32, 4);
dim3 blockPerGrid_ew_32(2, (j_size-2+3)/4);

dim3 threadPerBlock_sn_32(32, 4);
dim3 blockPerGrid_sn_32((i_size-2+31)/32, 16);

dim3 threadPerBlock_ew_b1(1, 128);
dim3 blockPerGrid_ew_b1(1, (j_size+127)/128);

dim3 threadPerBlock_ew_b2(1, 128);
dim3 blockPerGrid_ew_b2(2, (j_size+127)/128);

dim3 threadPerBlock_sn_b1(128, 1);
dim3 blockPerGrid_sn_b1((i_size+127)/128, 1);

dim3 threadPerBlock_sn_b2(128, 1);
dim3 blockPerGrid_sn_b2((i_size+127)/128, 2);


/*
//FILE *out;

//persistent variables
//1D const

float *d_dzz, *d_dz, *d_zz, *d_z;
int *d_j_global;
float *d_uabe, *d_uabw, *d_ele, *d_elw;
float *d_vabs, *d_vabn, *d_els, *d_eln;

float *d_aam_aid;

//2D const
float *d_fsm, *d_aru, *d_arv, *d_art;
float *d_dx, *d_dy, *d_dum, *d_dvm;
float *d_cor, *d_cbc,  *d_h;
float *d_tsurf, *d_ssurf;
float *d_tbe, *d_sbe, *d_tbw, *d_sbw;
float *d_tbs, *d_sbs, *d_tbn, *d_sbn;

//3D const
float *d_rmean, *d_tclim, *d_sclim;
/////////////////////////////////////////////////////

float *d_dt;
float *d_etf, *d_aam;



float *d_u, *d_v, *d_etb, *d_w;//added by advt2
float *d_advua, *d_advva, *d_d, *d_ua, *d_va, 
 	  *d_fluxua, *d_fluxva, *d_uab, *d_vab, *d_aam2d;//added by advave
float *d_advx, *d_advy, *d_ub, *d_vb; //added by advct
float *d_egf, *d_egb, *d_e_atmos, *d_drhox, *d_uf;//added by advu
float *d_drhoy, *d_vf;//added by advv
float *d_rho;//added by baropg 
float *d_vfluxf, *d_vfluxb;//added by vertvl
float *d_kq, *d_wusurf, *d_wvsurf, *d_wubot, *d_wvbot;//added by profq
float *d_t, *d_s, *d_q2b, *d_q2lb, *d_l, *d_km, *d_kh, *d_q2;//added by profq
float *d_swrad;//added by proft
float *d_wtsurf, *d_wssurf;//added by surface_forcing 
float *d_el;//added by mode_interaction
float *d_elb;//added by mode_external
float *d_utb, *d_vtb, *d_q2l;//added by mode_internal
float *d_tb, *d_sb, *d_et; //added by mode_internal

/////////////////////////////////////////////////////
//variable
//need not copy-in
float *d_adx2d, *d_ady2d, *d_drx2d, *d_dry2d; //added by mode_interaction;
float *d_elf, *d_utf, *d_vtf; //added by mode_external
float *d_uaf, *d_vaf; //added by mode_external
*/



/*
//local variables
float *d_3d_tmp0, *d_3d_tmp1, *d_3d_tmp2, 
	  *d_3d_tmp3, *d_3d_tmp4, *d_3d_tmp5,
	  *d_3d_tmp6; //*d_3d_tmp7, *d_3d_tmp8,
	  //*d_3d_tmp9, *d_3d_tmp10, *d_3d_tmp11,
	  //*d_3d_tmp12;


//double *d_3d_tmp0_d, *d_3d_tmp1_d, 
//	   *d_3d_tmp2_d, *d_3d_tmp3_d;


float *d_2d_tmp0, *d_2d_tmp1, *d_2d_tmp2; 
*/

void check(hipError_t err, const char* file, const char* func, unsigned line){
	if (err != hipSuccess){
		fprintf(stderr, "Hello: ERR: %s, file is %s, func is %s, line is %u, my_task is %d\n", 
				hipGetErrorString(err), file, func, line, my_task);	
		exit(1);
	}
}

void init_cuda_scalar_const_(int n_east, int n_west, int n_north, int n_south,
							 int my_task, int kb, int jm, int im,
							 int nitera, int mode, int ntp,
							 float sw, float dit2, float tprni, float grav, 
							 float tbias, float sbias, float rhoref, float umol,
							 float kappa, float small){

	return;
}


void init_cuda_1d_const(){
	
	checkCudaErrors(hipMalloc((void**) &d_zz,  kb*sizeof(float)));

	//added by mode_interaction 
	checkCudaErrors(hipMalloc((void**) &d_dz,  kb*sizeof(float)));

	//added by mode_external
	checkCudaErrors(hipMalloc((void**) &d_uabe, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uabw, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ele, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elw, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vabs, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vabn, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_els, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_eln, im*sizeof(float)));

	//added by mode_internal--profq
	checkCudaErrors(hipMalloc((void**) &d_dzz,  kb*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_z, kb*sizeof(float)));
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	checkCudaErrors(hipMemcpy(d_zz,  zz,  kb*sizeof(float), 
							   hipMemcpyHostToDevice));

	//added by mode_interaction 
	checkCudaErrors(hipMemcpy(d_dz,  dz,  kb*sizeof(float), 
							   hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_uabe, uabe, jm*sizeof(float), 
						       hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uabw, uabw, jm*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ele, ele, jm*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_elw, elw, jm*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vabs, vabs, im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vabn, vabn, im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_els, els, im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_eln, eln, im*sizeof(float), 
							   hipMemcpyHostToDevice));


	//added by mode_internal--profq
	checkCudaErrors(hipMemcpy(d_z,   z,   kb*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dzz, dzz, kb*sizeof(float), 
							   hipMemcpyHostToDevice));
	/*
	checkCudaErrors(hipMalloc((void**) &d_z,   kb*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_zz,  kb*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dz,  kb*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dzz, kb*sizeof(float)));

	checkCudaErrors(hipMalloc((void**) &d_j_global, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uabe, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uabw, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ele, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elw, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vabs, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vabn, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_els, im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_eln, im*sizeof(float)));


	checkCudaErrors(hipMemcpy(d_z,   z,   kb*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_zz,  zz,  kb*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dz,  dz,  kb*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dzz, dzz, kb*sizeof(float), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_j_global, j_global, jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uabe, uabe, jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uabw, uabw, jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ele, ele, jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_elw, elw, jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vabs, vabs, im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vabn, vabn, im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_els, els, im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_eln, eln, im*sizeof(float), hipMemcpyHostToDevice));

	initial_constant_csolver(dzz, dz, zz, z);
	*/
	return;

}

/*
void init_cuda_2d_const(float* aru, float* arv, float *art,
						 float *dx, float *dy,
						 float *dum, float *dvm, 
						 float* h, float *cor, 
						 float* fsm, float *cbc,
						 float *tsurf, float *ssurf,
						 float *tbe, float *sbe,
						 float *tbw, float *sbw,
						 float *tbs, float *sbs,
						 float *tbn, float *sbn){
*/
void init_cuda_2d_const(){

	//added by momentum3d
	checkCudaErrors(hipMalloc((void**) &d_dx,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dy,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_aamfrz,  jm*im*sizeof(float)));
	//added by momentum3d-->advct
	checkCudaErrors(hipMalloc((void**) &d_aru, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_arv, jm*im*sizeof(float)));
	//added by mode_external
	checkCudaErrors(hipMalloc((void**) &d_art, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cor, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_h, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_fsm, jm*im*sizeof(float)));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMalloc((void**) &d_frz, jm*im*sizeof(float)));




	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////


	checkCudaErrors(hipMemcpy(d_dx,  dx,  jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dy,  dy,  jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aamfrz,  aamfrz,  jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));
	//added by momentum3d-->advct
	checkCudaErrors(hipMemcpy(d_aru, aru, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_arv, arv, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));
	//added by mode_external
	checkCudaErrors(hipMemcpy(d_art, art, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cor, cor, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_h, h, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fsm, fsm, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMemcpy(d_frz, frz, jm*im*sizeof(float), 
								hipMemcpyHostToDevice));



	/*
	//below are for persistent global variables memroy alloc
	checkCudaErrors(hipMalloc((void**) &d_aru, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_arv, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_art, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dx,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dy,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dum, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dvm, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_h,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cor, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_fsm, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cbc, jm*im*sizeof(float)));

	checkCudaErrors(hipMalloc((void**) &d_tsurf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ssurf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbe, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbe, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbw, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbw, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbs, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbs, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbn, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbn, kb*im*sizeof(float)));
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	checkCudaErrors(hipMemcpy(d_aru, aru, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_arv, arv, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_art, art, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dx,  dx,  jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dy,  dy,  jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dum, dum, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dvm, dvm, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_h,   h,   jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cor, cor, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fsm, fsm, jm*im*sizeof(float), hipMemcpyHostToDevice));


	checkCudaErrors(hipMemcpy(d_cbc, cbc, jm*im*sizeof(float), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_tsurf, tsurf, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ssurf, ssurf, jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbe, tbe, kb*jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbe, sbe, kb*jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbw, tbw, kb*jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbw, sbw, kb*jm*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbs, tbs, kb*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbs, sbs, kb*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbn, tbn, kb*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbn, sbn, kb*im*sizeof(float), hipMemcpyHostToDevice));
	*/

	return;
}


void init_cuda_3d_const(){

	//added by mode_internal-->advt1
	checkCudaErrors(hipMalloc((void**) &d_tclim, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sclim, kb*jm*im*sizeof(float)));

	//added by mode_internal-->advt1
	checkCudaErrors(hipMalloc((void**) &d_relax_aid, kb*jm*im*sizeof(float)));

	////////////////////////////////////////////////

	checkCudaErrors(hipMemcpy(d_tclim, tclim, kb*jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sclim, sclim, kb*jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));


	//added by mode_internal-->advt1
	checkCudaErrors(hipMemcpy(d_relax_aid, relax_aid, 
							   kb*jm*im*sizeof(float), 
							   hipMemcpyHostToDevice));
	/*
	checkCudaErrors(hipMalloc((void**) &d_rmean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tclim, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sclim, kb*jm*im*sizeof(float)));
	////////////////////////////////////////////////
	//kq is a special variable, because it is only referenced and modified in profq
	//but we have to make it a global variable for the value is useful in the next iteration
	////////////////////////////////////////////////
	checkCudaErrors(hipMalloc((void**) &d_kq,    kb*jm*im*sizeof(float)));

	checkCudaErrors(hipMemcpy(d_rmean, rmean, kb*jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tclim, tclim, kb*jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sclim, sclim, kb*jm*im*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_kq,    kq,    kb*jm*im*sizeof(float), hipMemcpyHostToDevice));
	*/

	return;
}


void init_cuda_1d_var(){

	/*
	checkCudaErrors(hipMalloc((void**) &d_aam_aid, jm*sizeof(float)));

	checkCudaErrors(hipMemcpy(d_aam_aid, aam_aid, jm*sizeof(float), hipMemcpyHostToDevice));
	*/
	return;
}

/*
void init_cuda_2d_var(float *dt, float *el,
					  float *ua, float *va,
					  float *d, float *uab, 
					  float *vab, float *aam2d,
					  float *wubot, float *wvbot, 
					  float *advua, float *advva,
					  float *adx2d, float *ady2d,
					  float *drx2d, float *dry2d,
					  float *elb, float *etf,
					  float *utb, float *vtb,
					  float *etb, float *vfluxb,
					  float *egb, float *et){
*/
void init_cuda_2d_var(){

	//added by surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_vfluxf, 
							   jm*im*sizeof(float)));
	//assigned 0 in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_e_atmos, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_swrad, 
							   jm*im*sizeof(float)));

	//added by momentum3d
	checkCudaErrors(hipMalloc((void**) &d_dum, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dvm, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dt, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_d, 
							   jm*im*sizeof(float)));
	//added by mode_interaction 
	checkCudaErrors(hipMalloc((void**) &d_adx2d, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ady2d, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drx2d, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dry2d, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_aam2d, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ua, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_va, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uab, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vab, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wubot, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wvbot, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advua, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advva, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_el, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_egf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_utf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vtf, 
							   jm*im*sizeof(float)));
	//added by mode_interaction --advave
	checkCudaErrors(hipMalloc((void**) &d_cbc, 
							   jm*im*sizeof(float)));

	//added by mode_external
	checkCudaErrors(hipMalloc((void**) &d_elb, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wusurf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wvsurf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_etf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uaf, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vaf, 
							   jm*im*sizeof(float)));

	//added by mode_internal
	checkCudaErrors(hipMalloc((void**) &d_utb, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vtb, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_etb, 
							   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vfluxb, 
							   jm*im*sizeof(float)));

	//added by mode_internal-->advt1
	checkCudaErrors(hipMalloc((void**) &d_tsurf, jm*im*sizeof(float)));

	//added by mode_internal-->proft
	checkCudaErrors(hipMalloc((void**) &d_wtsurf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wssurf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ssurf, jm*im*sizeof(float)));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMalloc((void**) &d_tbe, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbe, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbw, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbw, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbs, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbs, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tbn, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sbn, kb*im*sizeof(float)));

	//added by mode_internal-->advu
	checkCudaErrors(hipMalloc((void**) &d_egb, jm*im*sizeof(float)));

	//added by mode_internal-->kernel_4
	checkCudaErrors(hipMalloc((void**) &d_et, jm*im*sizeof(float)));

	//added by store_mean
	checkCudaErrors(hipMalloc((void**) &d_uab_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vab_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elb_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wusurf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wvsurf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wtsurf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wssurf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_usrf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vsrf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elsrf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uwsrf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vwsrf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_utf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vtf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uwsrf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vwsrf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_celg_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ctsurf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_celg, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ctsurf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cpvf_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cjbar_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cpvf, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cjbar, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cadv_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cten_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cadv, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_cten, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ctbot_mean, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ctbot, jm*im*sizeof(float)));

	//added by mode_external-->vort 
	checkCudaErrors(hipMalloc((void**) &d_ctot, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_totx, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_toty, jm*im*sizeof(float)));

    /////////////////////////////////////////////////////

	checkCudaErrors(hipMemcpy(d_vfluxf, vfluxf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dum, dum, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dvm, dvm, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dt, dt, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_d, d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_adx2d, adx2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ady2d, ady2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_drx2d, drx2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dry2d, dry2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aam2d, aam2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ua, ua, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_va, va, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uab, uab, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vab, vab, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wubot, wubot, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wvbot, wvbot, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advua, advua, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advva, advva, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_el, el, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cbc, cbc, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_elb, elb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wusurf, wusurf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wvsurf, wvsurf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_etf, etf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));


	//added by mode_internal
	checkCudaErrors(hipMemcpy(d_utb, utb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vtb, vtb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_etb, etb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vfluxb, vfluxb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_internal-->proft
	checkCudaErrors(hipMemcpy(d_wtsurf, wtsurf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wssurf, wssurf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMemcpy(d_tbe, tbe, kb*jm*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbe, sbe, kb*jm*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbw, tbw, kb*jm*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbw, sbw, kb*jm*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbs, tbs, kb*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbs, sbs, kb*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tbn, tbn, kb*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sbn, sbn, kb*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_internal-->advu
	checkCudaErrors(hipMemcpy(d_egb, egb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_internal-->kernel_4
	checkCudaErrors(hipMemcpy(d_et, et, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////

	checkCudaErrors(hipMemset(d_e_atmos, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_swrad, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_egf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_utf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vtf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_elf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_uaf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vaf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_tsurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ssurf, 0, jm*im*sizeof(float)));

	//added by store_mean
	checkCudaErrors(hipMemset(d_uab_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vab_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_elb_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wusurf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wvsurf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wtsurf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wssurf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_usrf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vsrf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_elsrf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_uwsrf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vwsrf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_celg_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ctsurf_mean, 0, jm*im*sizeof(float)));

	checkCudaErrors(hipMemset(d_celg, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ctsurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cpvf_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cjbar_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cpvf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cjbar, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cadv_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cten_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cadv, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_cten, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ctbot_mean, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ctbot, 0, jm*im*sizeof(float)));

	//added by mode_external-->vort
	checkCudaErrors(hipMemset(d_ctot, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_totx, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_toty, 0, jm*im*sizeof(float)));

	/*
	checkCudaErrors(hipMalloc((void**) &d_dt,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_el,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ua,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_va,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_d,       jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uab,     jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vab,     jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_aam2d,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wubot,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_wvbot,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advua,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advva,   jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_adx2d,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ady2d,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drx2d,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_dry2d,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elb,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_etf,     jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_utb,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vtb,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_etb,     jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vfluxb,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_egb,     jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_et,      jm*im*sizeof(float)));



    /////////////////////////////////////////////////////
	//variable
	//need not copy-in
	//egf need not copied in for it is assigned in mode_interaction
	checkCudaErrors(hipMalloc((void**) &d_utf,      jm*im*sizeof(float)));
	//utf need not copied in for it is assigned in mode_interaction
	checkCudaErrors(hipMalloc((void**) &d_egf,     jm*im*sizeof(float)));
	//vtf need not copied in for it is assigned in mode_interaction
	checkCudaErrors(hipMalloc((void**) &d_vtf,      jm*im*sizeof(float)));
	//vfluxf need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_vfluxf,  jm*im*sizeof(float)));
	//e_atmos need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_e_atmos, jm*im*sizeof(float)));
	//wusurf need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_wusurf,  jm*im*sizeof(float)));
	//wvsurf need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_wvsurf,  jm*im*sizeof(float)));
	//wtsurf need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_wtsurf,      jm*im*sizeof(float)));
	//wssurf need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_wssurf,      jm*im*sizeof(float)));
	//swrad need not copied in for it is assigned in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_swrad,   jm*im*sizeof(float)));

	//checkCudaErrors(hipMalloc((void**) &d_dry2d,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_fluxua,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_fluxva,  jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_elf,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uaf,      jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vaf,      jm*im*sizeof(float)));



	//added by surface forcing
	checkCudaErrors(hipMemcpy(d_dt, dt, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	//added by mode_interaction
	checkCudaErrors(hipMemcpy(d_el, el, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ua, ua, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_va, va, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_d, d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uab, uab, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vab, vab, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aam2d, aam2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wubot, wubot, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wvbot, wvbot, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advua, advua, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advva, advva, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	//comment: below are only assigned 0, but in iopnetcdf,
	//they are read, I am not sure whether below are useful,
	//now I just read them;
	checkCudaErrors(hipMemcpy(d_adx2d, adx2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ady2d, ady2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_drx2d, drx2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_dry2d, dry2d, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_external
	checkCudaErrors(hipMemcpy(d_elb, elb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_etf, etf, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by mode_internal
	checkCudaErrors(hipMemcpy(d_utb, utb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vtb, vtb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_etb, etb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vfluxb, vfluxb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_egb, egb, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_et, et, jm*im*sizeof(float), 
					hipMemcpyHostToDevice));

	//added by surface-forcing
	checkCudaErrors(hipMemset(d_wusurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wvsurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wtsurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_wssurf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_e_atmos, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_swrad, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vfluxf, 0, jm*im*sizeof(float)));

	checkCudaErrors(hipMemset(d_utf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_egf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vtf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_fluxua, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_fluxva, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_elf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_uaf, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_vaf, 0, jm*im*sizeof(float)));
	*/

	return;
}


/*
void init_cuda_3d_var(float *t, float *s,
					  float *u, float *v,
					  float *ub, float *vb,
					  float *aam, float *rho,
					  float *advx, float *advy,
					  float *drhox, float *drhoy,
					  float *q2b, float *q2,
					  float *q2lb, float *q2l,
					  float *tb, float *sb, 
					  float *km, float *kh){
*/
void init_cuda_3d_var(){

	//re_assigned vfluxf in surface_forcing
	checkCudaErrors(hipMalloc((void**) &d_w,     kb*jm*im*sizeof(float)));

	//added by momentum3d
	checkCudaErrors(hipMalloc((void**) &d_u,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_v,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ub,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vb,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vb,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_aam,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_rho,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_rmean,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advx,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advy,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drhox,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drhoy,     kb*jm*im*sizeof(float)));

	//added by mode_internal
	checkCudaErrors(hipMalloc((void**) &d_q2b,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2lb,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2l,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_uf,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vf,     kb*jm*im*sizeof(float)));

	//added by mode_internal-->profq
	checkCudaErrors(hipMalloc((void**) &d_kq,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_l,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_t,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_s,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_km,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_kh,     kb*jm*im*sizeof(float)));
	//added by mode_internal-->advt1
	checkCudaErrors(hipMalloc((void**) &d_tb,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sb,     kb*jm*im*sizeof(float)));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMalloc((void**) &d_tobw,   kb*jm*nfw*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sobw,   kb*jm*nfw*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tobe,   kb*jm*nfe*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sobe,   kb*jm*nfe*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tobs,   kb*nfs*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sobs,   kb*nfs*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tobn,   kb*nfn*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sobn,   kb*nfn*im*sizeof(float)));

	//added by store_mean
	checkCudaErrors(hipMalloc((void**) &d_u_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_v_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_w_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_t_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_s_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_rho_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_kh_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_km_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ustks, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vstks, kb*jm*im*sizeof(float)));

	checkCudaErrors(hipMalloc((void**) &d_xstks_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ystks_mean, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_xstks, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ystks, kb*jm*im*sizeof(float)));


    /////////////////////////////////////////////////////
	//added by momentum3d
	checkCudaErrors(hipMemcpy(d_u, u, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_v, v, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ub, ub, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vb, vb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aam, aam, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_rho, rho, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_rmean, rmean, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by mode_internal
	checkCudaErrors(hipMemcpy(d_q2b, q2b, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_q2, q2, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_q2lb, q2lb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_q2l, q2l, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uf, uf, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vf, vf, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by mode_internal-->profq
	checkCudaErrors(hipMemcpy(d_kq, kq, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_t, t, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_s, s, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_km, km, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_kh, kh, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by mode_internal-->advt1
	checkCudaErrors(hipMemcpy(d_tb, tb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sb, sb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by mode_internal-->bcond(4)
	checkCudaErrors(hipMemcpy(d_tobw, tobw, kb*jm*nfw*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sobw, sobw, kb*jm*nfw*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tobe, tobe, kb*jm*nfe*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sobe, sobe, kb*jm*nfe*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tobs, tobs, kb*nfs*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sobs, sobs, kb*nfs*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tobn, tobn, kb*nfn*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sobn, sobn, kb*nfn*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by store_mean
	checkCudaErrors(hipMemcpy(d_ustks, ustks, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vstks, vstks, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_xstks, xstks, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ystks, ystks, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////
	checkCudaErrors(hipMemset(d_w, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_advx, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_advy, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_drhox, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_drhoy, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_l, 0, kb*jm*im*sizeof(float)));

	checkCudaErrors(hipMemset(d_u_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_v_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_w_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_t_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_s_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_rho_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_kh_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_km_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_xstks_mean, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_ystks_mean, 0, kb*jm*im*sizeof(float)));

	/*
	//below is for global variables memory alloc
	checkCudaErrors(hipMalloc((void**) &d_t,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_s,	  kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_u,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_v,     kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_ub,    kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_vb,    kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_aam,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_rho,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advx,  kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_advy,  kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drhox, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_drhoy, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2b,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2,    kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2lb,  kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_q2l,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_tb,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_sb,   kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_km,	  kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_kh,	  kb*jm*im*sizeof(float)));


	//added by surface forcing
	//in fact we just use s[0] and t[0],
	//what's worse, we multply them by 0...
	//but mode_internal will use whole s and t 
	checkCudaErrors(hipMemcpy(d_t, t, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_s, s, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by surface forcing
	checkCudaErrors(hipMemcpy(d_u, u, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_v, v, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ub, ub, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vb, vb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aam, aam, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));

	//added by surface forcing
	checkCudaErrors(hipMemcpy(d_rho, rho, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advx, advx, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_advy, advy, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_drhox, drhox, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_drhoy, drhoy, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)

	//added by mode_internal
	checkCudaErrors(hipMemcpy(d_q2b, q2b, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_q2, q2, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_q2lb, q2lb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_q2l, q2l, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_tb, tb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_sb, sb, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_km, km, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)
	checkCudaErrors(hipMemcpy(d_kh, kh, kb*jm*im*sizeof(float), 
				hipMemcpyHostToDevice));//it is assigned in initial.f(baropg_f)

	*/
	
	return;
}


void init_cuda_local(){


	checkCudaErrors(hipMalloc((void**) &d_3d_tmp0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp1, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp2, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp3, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp4, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp5, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_3d_tmp6, kb*jm*im*sizeof(float)));

	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp7, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp8, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp9, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp10, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp11, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMalloc((void**) &d_3d_tmp12, kb*jm*im*sizeof(float)));

	checkCudaErrors(hipMalloc((void**) &d_2d_tmp0, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp1, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp2, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp3, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp4, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp5, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp6, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp7, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp8, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp9, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp10, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp11, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp12, jm*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_tmp13, jm*im*sizeof(float)));

	//for data exchange in exchange2d_mpi
	checkCudaErrors(hipMalloc((void**) &d_1d_ny_tmp0, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_1d_ny_tmp1, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_1d_ny_tmp2, jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_1d_ny_tmp3, jm*sizeof(float)));

	//for data exchange in exchange3d_mpi
	checkCudaErrors(hipMalloc((void**) &d_2d_ny_nz_tmp0, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_ny_nz_tmp1, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_ny_nz_tmp2, kb*jm*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_ny_nz_tmp3, kb*jm*sizeof(float)));

	checkCudaErrors(hipMalloc((void**) &d_2d_nx_nz_tmp0, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_nx_nz_tmp1, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_nx_nz_tmp2, kb*im*sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &d_2d_nx_nz_tmp3, kb*im*sizeof(float)));

	checkCudaErrors(hipStreamCreate(&stream[0]));	
	for (int i = 1; i < 5; i++){
		checkCudaErrors(hipStreamCreateWithPriority(
						&stream[i], hipStreamDefault, -1));	
	}

	checkCudaErrors(hipMemset(d_3d_tmp0, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp1, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp2, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp3, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp4, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp5, 0, kb*jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_3d_tmp6, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp7, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp8, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp9, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp10, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp11, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_3d_tmp12, 0, kb*jm*im*sizeof(float)));

	checkCudaErrors(hipMemset(d_2d_tmp0, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp1, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp2, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp3, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp4, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp5, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp6, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp7, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp8, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp9, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp10, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp11, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp12, 0, jm*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_tmp13, 0, jm*im*sizeof(float)));


	checkCudaErrors(hipMemset(d_1d_ny_tmp0, 0, jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_1d_ny_tmp1, 0, jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_1d_ny_tmp2, 0, jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_1d_ny_tmp3, 0, jm*sizeof(float)));

	checkCudaErrors(hipMemset(d_2d_ny_nz_tmp0, 0, kb*jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_ny_nz_tmp1, 0, kb*jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_ny_nz_tmp2, 0, kb*jm*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_ny_nz_tmp3, 0, kb*jm*sizeof(float)));

	checkCudaErrors(hipMemset(d_2d_nx_nz_tmp0, 0, kb*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_nx_nz_tmp1, 0, kb*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_nx_nz_tmp2, 0, kb*im*sizeof(float)));
	checkCudaErrors(hipMemset(d_2d_nx_nz_tmp3, 0, kb*im*sizeof(float)));


	//checkCudaErrors(hipMemset(d_3d_tmp0_d, 0, kb*jm*im*sizeof(double)));
	//checkCudaErrors(hipMemset(d_3d_tmp1_d, 0, kb*jm*im*sizeof(double)));
	//checkCudaErrors(hipMemset(d_3d_tmp2_d, 0, kb*jm*im*sizeof(double)));
	//checkCudaErrors(hipMemset(d_3d_tmp3_d, 0, kb*jm*im*sizeof(double)));

	//checkCudaErrors(hipMemset(d_uf, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_vf, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_w, 0, kb*jm*im*sizeof(float)));
	//checkCudaErrors(hipMemset(d_l, 0, kb*jm*im*sizeof(float)));

	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	return;
}


//void init_cuda_ipc(){
void exchangeMemHandle(){
	checkCudaErrors(hipIpcGetMemHandle(&handle_ctsurf, (void*)d_ctsurf));
	checkCudaErrors(hipIpcGetMemHandle(&handle_ctbot, (void*)d_ctbot));
	checkCudaErrors(hipIpcGetMemHandle(&handle_celg, (void*)d_celg));
	checkCudaErrors(hipIpcGetMemHandle(&handle_cjbar, (void*)d_cjbar));
	checkCudaErrors(hipIpcGetMemHandle(&handle_cadv, (void*)d_cadv));
	checkCudaErrors(hipIpcGetMemHandle(&handle_cpvf, (void*)d_cpvf));
	checkCudaErrors(hipIpcGetMemHandle(&handle_cten, (void*)d_cten));


	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp0, (void*)d_2d_tmp0));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp1, (void*)d_2d_tmp1));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp2, (void*)d_2d_tmp2));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp3, (void*)d_2d_tmp3));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp4, (void*)d_2d_tmp4));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp5, (void*)d_2d_tmp5));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp6, (void*)d_2d_tmp6));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp7, (void*)d_2d_tmp7));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp8, (void*)d_2d_tmp8));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp9, (void*)d_2d_tmp9));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp10, (void*)d_2d_tmp10));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp11, (void*)d_2d_tmp11));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp12, (void*)d_2d_tmp12));
	checkCudaErrors(hipIpcGetMemHandle(&handle_2d_tmp13, (void*)d_2d_tmp13));

	checkCudaErrors(hipIpcGetMemHandle(&handle_totx, (void*)d_totx));
	checkCudaErrors(hipIpcGetMemHandle(&handle_toty, (void*)d_toty));

	checkCudaErrors(hipIpcGetMemHandle(&handle_3d_tmp0, (void*)d_3d_tmp0));
	checkCudaErrors(hipIpcGetMemHandle(&handle_3d_tmp1, (void*)d_3d_tmp1));
	checkCudaErrors(hipIpcGetMemHandle(&handle_3d_tmp2, (void*)d_3d_tmp2));

	checkCudaErrors(hipIpcGetMemHandle(&handle_wubot, (void*)d_wubot));
	checkCudaErrors(hipIpcGetMemHandle(&handle_wvbot, (void*)d_wvbot));

	checkCudaErrors(hipIpcGetMemHandle(&handle_aam, (void*)d_aam));

	checkCudaErrors(hipIpcGetMemHandle(&handle_elf, (void*)d_elf));

	checkCudaErrors(hipIpcGetMemHandle(&handle_uaf, (void*)d_uaf));
	checkCudaErrors(hipIpcGetMemHandle(&handle_vaf, (void*)d_vaf));

	checkCudaErrors(hipIpcGetMemHandle(&handle_w, (void*)d_w));
	checkCudaErrors(hipIpcGetMemHandle(&handle_uf, (void*)d_uf));
	checkCudaErrors(hipIpcGetMemHandle(&handle_vf, (void*)d_vf));
	checkCudaErrors(hipIpcGetMemHandle(&handle_kh, (void*)d_kh));
	checkCudaErrors(hipIpcGetMemHandle(&handle_km, (void*)d_km));
	checkCudaErrors(hipIpcGetMemHandle(&handle_kq, (void*)d_kq));

	MPI_Status status;

	if (n_west != -1){
		MPI_Send(&handle_ctsurf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_ctbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_celg, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_cjbar, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_cadv, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_cpvf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_cten, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);


		MPI_Send(&handle_2d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp3, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp4, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp5, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp6, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp7, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp8, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp9, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp10, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp11, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp12, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_2d_tmp13, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);


		MPI_Send(&handle_totx, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_toty, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_3d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_3d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_3d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_aam, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		
		MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);
		MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_west, 0, pom_comm);

		//////////////////////////////////////////////////////////////

		MPI_Recv(&handle_ctsurf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_ctbot_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_celg_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_cjbar_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_cadv_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_cpvf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_cten_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);


		MPI_Recv(&handle_2d_tmp0_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp1_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp2_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp3_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp4_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp5_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp6_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp7_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp8_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp9_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp10_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp11_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp12_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp13_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);


		MPI_Recv(&handle_totx_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_toty_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);


		MPI_Recv(&handle_3d_tmp0_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_3d_tmp1_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_3d_tmp2_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_wubot_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_wvbot_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_aam_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_elf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_uaf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_vaf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_w_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);

		MPI_Recv(&handle_uf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
		MPI_Recv(&handle_vf_west, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_west, 0, pom_comm, &status);
	}


	if (n_east != -1){
		MPI_Recv(&handle_ctsurf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_ctbot_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_celg_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_cjbar_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_cadv_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_cpvf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_cten_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	


		MPI_Recv(&handle_2d_tmp0_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp1_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp2_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp3_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp4_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp5_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp6_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp7_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp8_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp9_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp10_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp11_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp12_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_2d_tmp13_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_totx_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_toty_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_3d_tmp0_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp1_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp2_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_wubot_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_wvbot_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_aam_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_elf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_uaf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_vaf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_w_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	

		MPI_Recv(&handle_uf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	
		MPI_Recv(&handle_vf_east, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_east, 0, pom_comm, &status);	


		MPI_Send(&handle_ctsurf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_ctbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_celg, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_cjbar, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_cadv, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_cpvf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_cten, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);



		MPI_Send(&handle_2d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp3, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp4, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp5, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp6, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp7, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp8, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp9, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp10, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp11, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp12, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_2d_tmp13, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);


		MPI_Send(&handle_totx, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_toty, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);


		MPI_Send(&handle_3d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_3d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_3d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_aam, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

		MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);
		MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_east, 0, pom_comm);

	}

	if (n_south != -1){

		MPI_Send(&handle_3d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_3d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_3d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_2d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_2d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		
		MPI_Send(&handle_aam, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);
		MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_south, 0, pom_comm);

		////////////////////////////////////////////////////////////////////

		MPI_Recv(&handle_3d_tmp0_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp1_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp2_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	

		MPI_Recv(&handle_2d_tmp0_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp1_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);

		MPI_Recv(&handle_wubot_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	
		MPI_Recv(&handle_wvbot_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	

		MPI_Recv(&handle_aam_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);

		MPI_Recv(&handle_elf_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	

		MPI_Recv(&handle_uaf_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);
		MPI_Recv(&handle_vaf_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);

		MPI_Recv(&handle_w_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	

		MPI_Recv(&handle_uf_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	
		MPI_Recv(&handle_vf_south, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_south, 0, pom_comm, &status);	
	}

	if (n_north != -1){

		MPI_Recv(&handle_3d_tmp0_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp1_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	
		MPI_Recv(&handle_3d_tmp2_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	

		MPI_Recv(&handle_2d_tmp0_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);
		MPI_Recv(&handle_2d_tmp1_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);
		
		MPI_Recv(&handle_wubot_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	
		MPI_Recv(&handle_wvbot_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	

		MPI_Recv(&handle_aam_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);

		MPI_Recv(&handle_elf_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	

		MPI_Recv(&handle_uaf_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);
		MPI_Recv(&handle_vaf_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);

		MPI_Recv(&handle_w_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	

		MPI_Recv(&handle_uf_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	
		MPI_Recv(&handle_vf_north, sizeof(hipIpcMemHandle_t), MPI_BYTE,
				 n_north, 0, pom_comm, &status);	

		////////////////////////////////////////////////////////////////////

		MPI_Send(&handle_3d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_3d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_3d_tmp2, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_2d_tmp0, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_2d_tmp1, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_aam, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);

		MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
		MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
				 n_north, 0, pom_comm);
	}

	if (!(n_east == -1 && n_west == -1)){
		int nproc_x = (im_global-2)/(im_local-2);
		printf("xsz_debug: nproc_x = %d, rank:%d\n", nproc_x, my_task);
		if (n_west == -1){
			MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_elf_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_uaf_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_vaf_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_w_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_uf_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_vf_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_kh, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_kh_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_km, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_km_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_kq, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_kq_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_wubot_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

			MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task+nproc_x-1, 0, pom_comm);
			MPI_Recv(&handle_wvbot_east_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task+nproc_x-1, 0, pom_comm, &status);

		}

		if (n_east == -1){
			MPI_Recv(&handle_elf_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_elf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_uaf_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_uaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_vaf_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_vaf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_w_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_w, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_uf_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_uf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_vf_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_vf, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_kh_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_kh, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_km_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_km, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_kq_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_kq, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_wubot_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_wubot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

			MPI_Recv(&handle_wvbot_west_most, sizeof(hipIpcMemHandle_t), 
					 MPI_BYTE, my_task-nproc_x+1, 0, pom_comm, &status);

			MPI_Send(&handle_wvbot, sizeof(hipIpcMemHandle_t), MPI_BYTE, 
					 my_task-nproc_x+1, 0, pom_comm);

		}

		//if (n_west == -1){
		//	int access;
		//	checkCudaErrors(hipDeviceCanAccessPeer(&access, my_task, 
		//											n_east));
		//	if (access){
		//		checkCudaErrors(hipDeviceEnablePeerAccess(n_east, 0));	
		//		printf("Enable Peer Access from %d to %d\n", 
		//			   my_task, n_east);
		//		checkCudaErrors(hipSetDevice(n_east));
		//		checkCudaErrors(hipDeviceEnablePeerAccess(my_task, 0));
		//		printf("Enable Peer Access from %d to %d\n", 
		//			   n_east, my_task);
		//		checkCudaErrors(hipSetDevice(my_task));
		//	}
		//}

		//if (n_east == -1){
		//	int access;	
		//	checkCudaErrors(hipDeviceCanAccessPeer(&access, my_task, 
		//											n_west));
		//	if (access){
		//		checkCudaErrors(hipDeviceEnablePeerAccess(n_west, 0));	
		//		printf("Enable Peer Access from %d to %d\n", 
		//			   my_task, n_west);
		//		checkCudaErrors(hipSetDevice(n_west));
		//		checkCudaErrors(hipDeviceEnablePeerAccess(my_task, 0));
		//		printf("Enable Peer Access from %d to %d\n", 
		//			   n_west, my_task);
		//		checkCudaErrors(hipSetDevice(my_task));
		//	}
		//}
	}

	printf("Exchange Memory Handle End!\n");
}

void openMemHandle(){

	printf("This program is now hard-coded for 1, 2, or 4 process\n");
	printf("For 2 and 4 process, There are 2 processes distributed in longitude\n");

	if (n_east != -1){
		//checkCudaErrors(hipSetDevice(n_east));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_ctsurf_east, handle_ctsurf_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_ctbot_east, handle_ctbot_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_celg_east, handle_celg_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cjbar_east, handle_cjbar_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cadv_east, handle_cadv_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cpvf_east, handle_cpvf_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cten_east, handle_cten_east,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp0_east, handle_2d_tmp0_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp1_east, handle_2d_tmp1_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp2_east, handle_2d_tmp2_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp3_east, handle_2d_tmp3_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp4_east, handle_2d_tmp4_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp5_east, handle_2d_tmp5_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp6_east, handle_2d_tmp6_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp7_east, handle_2d_tmp7_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp8_east, handle_2d_tmp8_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp9_east, handle_2d_tmp9_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp10_east, handle_2d_tmp10_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp11_east, handle_2d_tmp11_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp12_east, handle_2d_tmp12_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp13_east, handle_2d_tmp13_east,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_totx_east, handle_totx_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_toty_east, handle_toty_east,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp0_east, handle_3d_tmp0_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp1_east, handle_3d_tmp1_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp2_east, handle_3d_tmp2_east,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wubot_east, handle_wubot_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wvbot_east, handle_wvbot_east,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_aam_east, handle_aam_east,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_elf_east, handle_elf_east,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uaf_east, handle_uaf_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vaf_east, handle_vaf_east,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_w_east, handle_w_east,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uf_east, handle_uf_east,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vf_east, handle_vf_east,
							 hipIpcMemLazyEnablePeerAccess));

		//checkCudaErrors(hipSetDevice(my_task));
	}

	if (n_west != -1){

		//checkCudaErrors(hipSetDevice(n_west));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_ctsurf_west, handle_ctsurf_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_ctbot_west, handle_ctbot_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_celg_west, handle_celg_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cjbar_west, handle_cjbar_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cadv_west, handle_cadv_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cpvf_west, handle_cpvf_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_cten_west, handle_cten_west,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp0_west, handle_2d_tmp0_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp1_west, handle_2d_tmp1_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp2_west, handle_2d_tmp2_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp3_west, handle_2d_tmp3_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp4_west, handle_2d_tmp4_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp5_west, handle_2d_tmp5_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp6_west, handle_2d_tmp6_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp7_west, handle_2d_tmp7_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp8_west, handle_2d_tmp8_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp9_west, handle_2d_tmp9_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp10_west, handle_2d_tmp10_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp11_west, handle_2d_tmp11_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp12_west, handle_2d_tmp12_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp13_west, handle_2d_tmp13_west,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_totx_west, handle_totx_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_toty_west, handle_toty_west,
							 hipIpcMemLazyEnablePeerAccess));


		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp0_west, handle_3d_tmp0_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp1_west, handle_3d_tmp1_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp2_west, handle_3d_tmp2_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wubot_west, handle_wubot_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wvbot_west, handle_wvbot_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_aam_west, handle_aam_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_elf_west, handle_elf_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uaf_west, handle_uaf_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vaf_west, handle_vaf_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_w_west, handle_w_west,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uf_west, handle_uf_west,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vf_west, handle_vf_west,
							 hipIpcMemLazyEnablePeerAccess));

		//checkCudaErrors(hipSetDevice(my_task));
	}

	if (n_south != -1){
		checkCudaErrors(hipSetDevice(n_south));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp0_south, handle_3d_tmp0_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp1_south, handle_3d_tmp1_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp2_south, handle_3d_tmp2_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp0_south, handle_2d_tmp0_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp1_south, handle_2d_tmp1_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wubot_south, handle_wubot_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wvbot_south, handle_wvbot_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_aam_south, handle_aam_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_elf_south, handle_elf_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uaf_south, handle_uaf_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vaf_south, handle_vaf_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_w_south, handle_w_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uf_south, handle_uf_south,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vf_south, handle_vf_south,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipSetDevice(my_task));
	}

	if (n_north != -1){
		checkCudaErrors(hipSetDevice(n_north));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp0_north, handle_3d_tmp0_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp1_north, handle_3d_tmp1_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_3d_tmp2_north, handle_3d_tmp2_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp0_north, handle_2d_tmp0_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_2d_tmp1_north, handle_2d_tmp1_north,
							 hipIpcMemLazyEnablePeerAccess));
	
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wubot_north, handle_wubot_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_wvbot_north, handle_wvbot_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_aam_north, handle_aam_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_elf_north, handle_elf_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uaf_north, handle_uaf_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vaf_north, handle_vaf_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_w_north, handle_w_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_uf_north, handle_uf_north,
							 hipIpcMemLazyEnablePeerAccess));
		checkCudaErrors(hipIpcOpenMemHandle(
							(void**)&d_vf_north, handle_vf_north,
							 hipIpcMemLazyEnablePeerAccess));

		checkCudaErrors(hipSetDevice(my_task));
	}

	if (!(n_east == -1 && n_west == -1)){
		int nproc_x = (im_global-2)/(im_local-2);
		if (nproc_x == 2){
			if (n_west == -1){
				d_elf_east_most = d_elf_east;
				d_uaf_east_most = d_uaf_east;
				d_vaf_east_most = d_vaf_east;
				d_w_east_most = d_w_east;
				d_uf_east_most = d_uf_east;
				d_vf_east_most = d_vf_east;
				//d_kh_east_most = d_kh_east;
				//d_km_east_most = d_km_east;
				//d_kq_east_most = d_kq_east;
				d_wubot_east_most = d_wubot_east;
				d_wvbot_east_most = d_wvbot_east;

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kh_east_most, 
								handle_kh_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_km_east_most, 
								handle_km_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kq_east_most, 
								handle_kq_east_most,
								hipIpcMemLazyEnablePeerAccess));
			}
			if (n_east == -1){
				d_elf_west_most = d_elf_west;
				d_uaf_west_most = d_uaf_west;
				d_vaf_west_most = d_vaf_west;
				d_w_west_most = d_w_west;
				d_uf_west_most = d_uf_west;
				d_vf_west_most = d_vf_west;
				//d_kh_west_most = d_kh_west;
				//d_km_west_most = d_km_west;
				//d_kq_west_most = d_kq_west;
				d_wubot_west_most = d_wubot_west;
				d_wvbot_west_most = d_wvbot_west;

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kh_west_most, 
								handle_kh_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_km_west_most, 
								handle_km_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kq_west_most, 
								handle_kq_west_most,
								hipIpcMemLazyEnablePeerAccess));
			}
		}else{

			if (n_west == -1){
				//checkCudaErrors(hipSetDevice(my_task+nproc_x-1));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_elf_east_most, handle_elf_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_uaf_east_most, handle_uaf_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_vaf_east_most, handle_vaf_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_w_east_most, handle_w_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_uf_east_most, handle_uf_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_vf_east_most, handle_vf_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kh_east_most, handle_kh_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_km_east_most, handle_km_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kq_east_most, handle_kq_east_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_wubot_east_most, 
								handle_wubot_east_most,
								hipIpcMemLazyEnablePeerAccess));
				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_wvbot_east_most, 
								handle_wvbot_east_most,
								hipIpcMemLazyEnablePeerAccess));

				//checkCudaErrors(hipSetDevice(my_task));

			}

			if (n_east == -1){
				//checkCudaErrors(hipSetDevice(my_task-(nproc_x-1)));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_elf_west_most, handle_elf_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_uaf_west_most, handle_uaf_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_vaf_west_most, handle_vaf_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_w_west_most, handle_w_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_uf_west_most, handle_uf_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_vf_west_most, handle_vf_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kh_west_most, handle_kh_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_km_west_most, handle_km_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_kq_west_most, handle_kq_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_wubot_west_most, 
								handle_wubot_west_most,
								hipIpcMemLazyEnablePeerAccess));

				checkCudaErrors(hipIpcOpenMemHandle(
								(void**)&d_wvbot_west_most, 
								handle_wvbot_west_most,
								hipIpcMemLazyEnablePeerAccess));

				//checkCudaErrors(hipSetDevice(my_task));
			}
		}
	}
}

void init_cuda_peer(){

	///////////////////////////////////////////////////
	//This function is of no use, just to warm the hardware
	//we find a 1s latency when first calling hipMemcpyPeer
	//so we call it first in init phase
	//I think it is relevant to JIT
	
	hipMemcpy3DPeerParms p_east_recv={0};
	p_east_recv.extent = make_hipExtent(sizeof(float), jm, kb);
	p_east_recv.dstDevice = n_east;
	p_east_recv.dstPtr = make_hipPitchedPtr(d_3d_tmp0, im*sizeof(float), im, jm);
	p_east_recv.srcDevice = my_task;
	p_east_recv.srcPtr = make_hipPitchedPtr(d_3d_tmp1, im*sizeof(float), im, jm);

	hipMemcpy3DPeerParms p_west_recv={0};
	p_west_recv.extent = make_hipExtent(sizeof(float), jm, kb);
	p_west_recv.dstDevice = n_west;
	p_west_recv.dstPtr = make_hipPitchedPtr(d_3d_tmp0, im*sizeof(float), im, jm);
	p_west_recv.srcDevice = my_task;
	p_west_recv.srcPtr = make_hipPitchedPtr(d_3d_tmp1, im*sizeof(float), im, jm);

	if (n_east != -1){
		checkCudaErrors(hipMemcpy3DPeer(&p_east_recv));	
	}

	if (n_west != -1){
		checkCudaErrors(hipMemcpy3DPeer(&p_west_recv));	
	}
}

void init_cuda_pinned_memory(){
	//mpi
	checkCudaErrors(hipHostAlloc((void**)&h_1d_nx_tmp0, 
								  im*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_nx_tmp1, 
								  im*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_nx_tmp2, 
								  im*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_nx_tmp3, 
								  im*sizeof(float), 
								  hipHostMallocPortable));

	checkCudaErrors(hipHostAlloc((void**)&h_1d_ny_tmp0, 
								  jm*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_ny_tmp1, 
								  jm*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_ny_tmp2, 
								  jm*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_1d_ny_tmp3, 
								  jm*sizeof(float), 
								  hipHostMallocPortable));

	checkCudaErrors(hipHostAlloc((void**)&h_2d_nx_nz_tmp0, 
								  im*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_nx_nz_tmp1, 
								  im*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_nx_nz_tmp2, 
								  im*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_nx_nz_tmp3, 
								  im*kb*sizeof(float), 
								  hipHostMallocPortable));

	checkCudaErrors(hipHostAlloc((void**)&h_2d_ny_nz_tmp0, 
								  jm*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_ny_nz_tmp1, 
								  jm*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_ny_nz_tmp2, 
								  jm*kb*sizeof(float), 
								  hipHostMallocPortable));
	checkCudaErrors(hipHostAlloc((void**)&h_2d_ny_nz_tmp3, 
								  jm*kb*sizeof(float), 
								  hipHostMallocPortable));

	//print_section
	checkCudaErrors(hipHostRegister(tb, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(sb, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(ub, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(vb, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));


	checkCudaErrors(hipHostRegister(dt, jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(et, jm*im*sizeof(float), 
									 hipHostRegisterPortable));

	//output_copy_back
	checkCudaErrors(hipHostRegister(u, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(v, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(w, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(t, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(s, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(rho, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(kh, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(km, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));

	checkCudaErrors(hipHostRegister(uab, jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(vab, jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(elb, jm*im*sizeof(float), 
									 hipHostRegisterPortable));

	checkCudaErrors(hipHostRegister(u_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(v_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(w_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(t_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(s_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(rho_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(kh_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(km_mean, kb*jm*im*sizeof(float), 
									 hipHostRegisterPortable));

	checkCudaErrors(hipHostRegister(uab_mean, jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(vab_mean, jm*im*sizeof(float), 
									 hipHostRegisterPortable));
	checkCudaErrors(hipHostRegister(elb_mean, jm*im*sizeof(float), 
									 hipHostRegisterPortable));

	return;
}

void finalize_cuda_gpu(){
	//fclose(out);

	//1D const 
	checkCudaErrors(hipFree(d_zz));
	checkCudaErrors(hipFree(d_dz));
	checkCudaErrors(hipFree(d_uabe));
	checkCudaErrors(hipFree(d_uabw));
	checkCudaErrors(hipFree(d_ele));
	checkCudaErrors(hipFree(d_elw));
	checkCudaErrors(hipFree(d_vabs));
	checkCudaErrors(hipFree(d_vabn));
	checkCudaErrors(hipFree(d_els));
	checkCudaErrors(hipFree(d_eln));
	checkCudaErrors(hipFree(d_dzz));
	checkCudaErrors(hipFree(d_z));


	//checkCudaErrors(hipFree(d_j_global));
	//checkCudaErrors(hipFree(d_aam_aid));


	//2D const
	checkCudaErrors(hipFree(d_dx));
	checkCudaErrors(hipFree(d_dy));
	checkCudaErrors(hipFree(d_aru));
	checkCudaErrors(hipFree(d_arv));
	checkCudaErrors(hipFree(d_aamfrz));
	checkCudaErrors(hipFree(d_art));
	checkCudaErrors(hipFree(d_cor));
	checkCudaErrors(hipFree(d_h));
	checkCudaErrors(hipFree(d_fsm));
	checkCudaErrors(hipFree(d_frz));


	//3D const
	checkCudaErrors(hipFree(d_tclim));
	checkCudaErrors(hipFree(d_sclim));

	checkCudaErrors(hipFree(d_relax_aid));



	//2D
	checkCudaErrors(hipFree(d_vfluxf));
	checkCudaErrors(hipFree(d_e_atmos));
	checkCudaErrors(hipFree(d_swrad));
	checkCudaErrors(hipFree(d_dum));
	checkCudaErrors(hipFree(d_dvm));
	checkCudaErrors(hipFree(d_dt));
	checkCudaErrors(hipFree(d_d));
	checkCudaErrors(hipFree(d_adx2d));
	checkCudaErrors(hipFree(d_ady2d));
	checkCudaErrors(hipFree(d_drx2d));
	checkCudaErrors(hipFree(d_dry2d));
	checkCudaErrors(hipFree(d_aam2d));
	checkCudaErrors(hipFree(d_ua));
	checkCudaErrors(hipFree(d_va));
	checkCudaErrors(hipFree(d_uab));
	checkCudaErrors(hipFree(d_vab));
	checkCudaErrors(hipFree(d_wubot));
	checkCudaErrors(hipFree(d_wvbot));
	checkCudaErrors(hipFree(d_advua));
	checkCudaErrors(hipFree(d_advva));
	checkCudaErrors(hipFree(d_el));
	checkCudaErrors(hipFree(d_egf));
	checkCudaErrors(hipFree(d_utf));
	checkCudaErrors(hipFree(d_vtf));
	checkCudaErrors(hipFree(d_cbc));


	checkCudaErrors(hipFree(d_elb));
	checkCudaErrors(hipFree(d_elf));
	checkCudaErrors(hipFree(d_wusurf));
	checkCudaErrors(hipFree(d_wvsurf));
	checkCudaErrors(hipFree(d_etf));
	checkCudaErrors(hipFree(d_uaf));
	checkCudaErrors(hipFree(d_vaf));


	checkCudaErrors(hipFree(d_utb));
	checkCudaErrors(hipFree(d_vtb));
	checkCudaErrors(hipFree(d_etb));
	checkCudaErrors(hipFree(d_vfluxb));
	checkCudaErrors(hipFree(d_tsurf));
	checkCudaErrors(hipFree(d_wtsurf));
	checkCudaErrors(hipFree(d_wssurf));
	checkCudaErrors(hipFree(d_ssurf));


	checkCudaErrors(hipFree(d_tbe));
	checkCudaErrors(hipFree(d_sbe));
	checkCudaErrors(hipFree(d_tbw));
	checkCudaErrors(hipFree(d_sbw));
	checkCudaErrors(hipFree(d_tbs));
	checkCudaErrors(hipFree(d_sbs));
	checkCudaErrors(hipFree(d_tbn));
	checkCudaErrors(hipFree(d_sbn));


	checkCudaErrors(hipFree(d_egb));
	checkCudaErrors(hipFree(d_et));

	checkCudaErrors(hipFree(d_uab_mean));
	checkCudaErrors(hipFree(d_vab_mean));
	checkCudaErrors(hipFree(d_elb_mean));
	checkCudaErrors(hipFree(d_wusurf_mean));
	checkCudaErrors(hipFree(d_wvsurf_mean));
	checkCudaErrors(hipFree(d_wtsurf_mean));
	checkCudaErrors(hipFree(d_wssurf_mean));
	checkCudaErrors(hipFree(d_usrf_mean));
	checkCudaErrors(hipFree(d_vsrf_mean));
	checkCudaErrors(hipFree(d_elsrf_mean));
	checkCudaErrors(hipFree(d_uwsrf_mean));
	checkCudaErrors(hipFree(d_vwsrf_mean));
	checkCudaErrors(hipFree(d_uwsrf));
	checkCudaErrors(hipFree(d_vwsrf));
	checkCudaErrors(hipFree(d_utf_mean));
	checkCudaErrors(hipFree(d_vtf_mean));
	checkCudaErrors(hipFree(d_celg_mean));
	checkCudaErrors(hipFree(d_ctsurf_mean));
	checkCudaErrors(hipFree(d_celg));
	checkCudaErrors(hipFree(d_ctsurf));
	checkCudaErrors(hipFree(d_cpvf_mean));
	checkCudaErrors(hipFree(d_cjbar_mean));
	checkCudaErrors(hipFree(d_cpvf));
	checkCudaErrors(hipFree(d_cjbar));
	checkCudaErrors(hipFree(d_cadv_mean));
	checkCudaErrors(hipFree(d_cten_mean));
	checkCudaErrors(hipFree(d_cadv));
	checkCudaErrors(hipFree(d_cten));
	checkCudaErrors(hipFree(d_ctbot_mean));
	checkCudaErrors(hipFree(d_ctbot));


	//3D
	checkCudaErrors(hipFree(d_w));
	checkCudaErrors(hipFree(d_u));
	checkCudaErrors(hipFree(d_v));
	checkCudaErrors(hipFree(d_ub));
	checkCudaErrors(hipFree(d_vb));
	checkCudaErrors(hipFree(d_aam));
	checkCudaErrors(hipFree(d_rho));
	checkCudaErrors(hipFree(d_rmean));
	checkCudaErrors(hipFree(d_advx));
	checkCudaErrors(hipFree(d_advy));
	checkCudaErrors(hipFree(d_drhox));
	checkCudaErrors(hipFree(d_drhoy));


	checkCudaErrors(hipFree(d_q2b));
	checkCudaErrors(hipFree(d_q2));
	checkCudaErrors(hipFree(d_q2lb));
	checkCudaErrors(hipFree(d_q2l));
	checkCudaErrors(hipFree(d_uf));
	checkCudaErrors(hipFree(d_vf));
	checkCudaErrors(hipFree(d_kq));
	checkCudaErrors(hipFree(d_l));
	checkCudaErrors(hipFree(d_t));
	checkCudaErrors(hipFree(d_s));
	checkCudaErrors(hipFree(d_km));
	checkCudaErrors(hipFree(d_kh));
	checkCudaErrors(hipFree(d_tb));
	checkCudaErrors(hipFree(d_sb));

	checkCudaErrors(hipFree(d_tobw));
	checkCudaErrors(hipFree(d_sobw));
	checkCudaErrors(hipFree(d_tobe));
	checkCudaErrors(hipFree(d_sobe));
	checkCudaErrors(hipFree(d_tobs));
	checkCudaErrors(hipFree(d_sobs));
	checkCudaErrors(hipFree(d_tobn));
	checkCudaErrors(hipFree(d_sobn));


	checkCudaErrors(hipFree(d_u_mean));
	checkCudaErrors(hipFree(d_v_mean));
	checkCudaErrors(hipFree(d_w_mean));
	checkCudaErrors(hipFree(d_t_mean));
	checkCudaErrors(hipFree(d_s_mean));
	checkCudaErrors(hipFree(d_rho_mean));
	checkCudaErrors(hipFree(d_kh_mean));
	checkCudaErrors(hipFree(d_km_mean));
	checkCudaErrors(hipFree(d_ustks));
	checkCudaErrors(hipFree(d_vstks));

	checkCudaErrors(hipFree(d_xstks_mean));
	checkCudaErrors(hipFree(d_ystks_mean));
	checkCudaErrors(hipFree(d_xstks));
	checkCudaErrors(hipFree(d_ystks));

	
	checkCudaErrors(hipFree(d_3d_tmp0));
	checkCudaErrors(hipFree(d_3d_tmp1));
	checkCudaErrors(hipFree(d_3d_tmp2));
	checkCudaErrors(hipFree(d_3d_tmp3));
	checkCudaErrors(hipFree(d_3d_tmp4));
	checkCudaErrors(hipFree(d_3d_tmp5));
	checkCudaErrors(hipFree(d_3d_tmp6));
	//checkCudaErrors(hipFree(d_3d_tmp7));
	//checkCudaErrors(hipFree(d_3d_tmp8));
	//checkCudaErrors(hipFree(d_3d_tmp9));
	//checkCudaErrors(hipFree(d_3d_tmp10));
	//checkCudaErrors(hipFree(d_3d_tmp11));
	//checkCudaErrors(hipFree(d_3d_tmp12));


	checkCudaErrors(hipFree(d_2d_tmp0));
	checkCudaErrors(hipFree(d_2d_tmp1));
	checkCudaErrors(hipFree(d_2d_tmp2));
	checkCudaErrors(hipFree(d_2d_tmp3));
	checkCudaErrors(hipFree(d_2d_tmp4));
	checkCudaErrors(hipFree(d_2d_tmp5));
	checkCudaErrors(hipFree(d_2d_tmp6));
	checkCudaErrors(hipFree(d_2d_tmp7));
	checkCudaErrors(hipFree(d_2d_tmp8));
	checkCudaErrors(hipFree(d_2d_tmp9));
	checkCudaErrors(hipFree(d_2d_tmp10));
	checkCudaErrors(hipFree(d_2d_tmp11));
	checkCudaErrors(hipFree(d_2d_tmp12));
	checkCudaErrors(hipFree(d_2d_tmp13));

	checkCudaErrors(hipFree(d_1d_ny_tmp0));
	checkCudaErrors(hipFree(d_1d_ny_tmp1));
	checkCudaErrors(hipFree(d_1d_ny_tmp2));
	checkCudaErrors(hipFree(d_1d_ny_tmp3));

	checkCudaErrors(hipFree(d_2d_ny_nz_tmp0));
	checkCudaErrors(hipFree(d_2d_ny_nz_tmp1));
	checkCudaErrors(hipFree(d_2d_ny_nz_tmp2));
	checkCudaErrors(hipFree(d_2d_ny_nz_tmp3));

	checkCudaErrors(hipFree(d_2d_nx_nz_tmp0));
	checkCudaErrors(hipFree(d_2d_nx_nz_tmp1));
	checkCudaErrors(hipFree(d_2d_nx_nz_tmp2));
	checkCudaErrors(hipFree(d_2d_nx_nz_tmp3));

	//free host alloc
	checkCudaErrors(hipHostFree(h_1d_nx_tmp0));
	checkCudaErrors(hipHostFree(h_1d_nx_tmp1));
	checkCudaErrors(hipHostFree(h_1d_nx_tmp2));
	checkCudaErrors(hipHostFree(h_1d_nx_tmp3));

	checkCudaErrors(hipHostFree(h_1d_ny_tmp0));
	checkCudaErrors(hipHostFree(h_1d_ny_tmp1));
	checkCudaErrors(hipHostFree(h_1d_ny_tmp2));
	checkCudaErrors(hipHostFree(h_1d_ny_tmp3));

	checkCudaErrors(hipHostFree(h_2d_nx_nz_tmp0));
	checkCudaErrors(hipHostFree(h_2d_nx_nz_tmp1));
	checkCudaErrors(hipHostFree(h_2d_nx_nz_tmp2));
	checkCudaErrors(hipHostFree(h_2d_nx_nz_tmp3));

	checkCudaErrors(hipHostFree(h_2d_ny_nz_tmp0));
	checkCudaErrors(hipHostFree(h_2d_ny_nz_tmp1));
	checkCudaErrors(hipHostFree(h_2d_ny_nz_tmp2));
	checkCudaErrors(hipHostFree(h_2d_ny_nz_tmp3));

	for (int i = 0; i < 5; i++){
		checkCudaErrors(hipStreamDestroy(stream[i]));	
	}
	//checkCudaErrors(hipDeviceReset());
	return; 
}

void finalize_cuda_ipc(){
	/*
	if (n_east != -1){
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_ctsurf_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_ctbot_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_celg_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cjbar_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cadv_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cpvf_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cten_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp0_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp1_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp2_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp3_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp4_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp5_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp6_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp7_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp8_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp9_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp10_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp11_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp12_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp13_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_totx_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_toty_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp0_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp1_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp2_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_wubot_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_wvbot_east));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_aam_east));
	}
	if (n_west != -1){
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_ctsurf_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_ctbot_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_celg_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cjbar_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cadv_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cpvf_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_cten_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp0_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp1_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp2_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp3_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp4_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp5_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp6_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp7_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp8_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp9_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp10_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp11_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp12_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_2d_tmp13_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_totx_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_toty_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp0_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp1_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_3d_tmp2_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_wubot_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_wvbot_west));
		checkCudaErrors(hipIpcCloseMemHandle((void*)d_aam_west));
	}
	*/
}

void init_device(){
#ifdef OPEN_MPI
	int local_rank = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK")); 
	int local_size = atoi(getenv("OMPI_COMM_WORLD_LOCAL_SIZE")); 
	printf("local_rank = %d, local_size = %d\n\n",
			local_rank, local_size);
	sleep(10);
	//hipSetDevice(local_rank%(local_size/2));
#endif
	
}

void init_device_impi(){
	const char *pciBus[4]={"0000:02:00.0", "0000:03:00.0",
						   "0000:83:00.0", "0000:84:00.0"};
	int deviceId;
	hipDeviceGetByPCIBusId(&deviceId, (char*)pciBus[(my_task)%4]);
	
	hipSetDevice(deviceId);
	printf("my_task %d setDevice %d\n", my_task, deviceId);
	sleep(10);
}


void end_device(){
	checkCudaErrors(hipDeviceReset());
}



